#include "hip/hip_runtime.h"
// nvcc cudnn_conv.cu -lcudnn -Xcompiler -fopenmp -O3 && ./a.out 100 100
#include "common.h"
#include <hipDNN.h>

int main(int argc, char **argv) {
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);

  float *x = alloc(batch * in_channel * in_size * in_size);
  float *w = alloc(out_channel * in_channel * kernel * kernel);
  float *y = alloc(batch * out_channel * out_size * out_size);
  float *y1 = alloc(batch * out_channel * out_size * out_size);

  for (int i = 0; i < batch * in_channel * in_size * in_size; ++i) x[i] = gen();
  for (int i = 0; i < out_channel * in_channel * kernel * kernel; ++i) w[i] = gen();

  float *d_x, *d_w, *d_y;
  hipMalloc((void **) &d_x, sizeof(float) * batch * in_channel * in_size * in_size);
  hipMalloc((void **) &d_w, sizeof(float) * out_channel * in_channel * kernel * kernel);
  hipMalloc((void **) &d_y, sizeof(float) * batch * out_channel * out_size * out_size);
  hipMemcpy(d_x, x, sizeof(float) * batch * in_channel * in_size * in_size, hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, sizeof(float) * out_channel * in_channel * kernel * kernel, hipMemcpyHostToDevice);

  hipdnnTensorDescriptor_t x_desc;
  hipdnnCreateTensorDescriptor(&x_desc);
  hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, in_channel, in_size, in_size);
  hipdnnFilterDescriptor_t w_desc;
  hipdnnCreateFilterDescriptor(&w_desc);
  hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, out_channel, in_channel, kernel, kernel);
  hipdnnTensorDescriptor_t y_desc;
  hipdnnCreateTensorDescriptor(&y_desc);
  hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, out_channel, out_size, out_size);

  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnCreateConvolutionDescriptor(&conv_desc);
  // stride = dilation = (1, 1)
  hipdnnSetConvolution2dDescriptor(conv_desc, pad, pad, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

  hipdnnConvolutionFwdAlgoPerf_t algo;
#ifdef ALGO
  algo.algo = ALGO;
#else
  int _;
  hipdnnFindConvolutionForwardAlgorithm(handle, x_desc, w_desc, conv_desc, y_desc, 1, &_, &algo);
#endif
  printf("algo = %d\n", (int) algo.algo);

  size_t workspace_size;
  hipdnnGetConvolutionForwardWorkspaceSize(handle, x_desc, w_desc, conv_desc, y_desc, algo.algo, &workspace_size);
  void *workspace;
  hipMalloc(&workspace, workspace_size);
  printf("workspace_size = %d, workspace = %p\n", (int) workspace_size, workspace);

  float alpha = 1, beta = 0;

  run(argc, argv, [=](int rep) {
    for (int i = 0; i < rep; ++i) {
      hipdnnConvolutionForward(handle, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, algo.algo,
                              workspace, workspace_size, &beta, y_desc, d_y);
      hipStreamSynchronize(0);
    }
  });

  conv_naive(x, w, y1);
  hipMemcpy(y, d_y, sizeof(float) * batch * out_channel * out_size * out_size, hipMemcpyDeviceToHost);
  print_diff(y, y1, batch * out_channel * out_size * out_size);
}
